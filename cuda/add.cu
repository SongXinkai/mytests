#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::endl;

#define CUDA_CHECK(x) \
    { hipError_t cuda_error = x; \
        if (cuda_error != hipSuccess) \
            cout << "cudaError_t: " << cuda_error << " != 0 " \
                 << hipGetErrorString(cuda_error) << endl; \
    }

#define VECTOR_PRINT(head_str, vec, len) \
    cout << head_str << ": {"; \
    for (int i = 0; i < len - 1; ++i){ \
        cout << vec[i] << ", "; \
    } \
    cout << vec[len - 1] << "}" << endl;

#define LEN 34

// kernel functions
template<typename Dtype>
__global__ void add_kernel(const int N, const Dtype* a, const Dtype* b, Dtype* c){
    int i = threadIdx.x; // thread index in block
    // c[i] = a[i] + b[i];
    
    for (int i = threadIdx.x; i < N; i += gridDim.x * blockDim.x){
        c[i] = a[i] + b[i];
    }
}

int main(){
    // host memory malloc & initial
    int* host_a = new int[LEN];
    int* host_b = new int[LEN];
    int* host_c = new int[LEN];
    for (int i = 0; i < LEN; ++i){
        host_a[i] = i;
        host_b[i] = i * 100;
        host_c[i] = -1;
    }
    
    // GPU device start
    int device_id = 1;
    CUDA_CHECK(hipSetDevice(device_id));
    cout << "Using GPU " << device_id << "." << endl;
    
    // cudaMalloc & cudaMemcpy & cudaMemset
    int* dev_a;
    int* dev_b;
    int* dev_c;
    CUDA_CHECK(hipMalloc((void**)&dev_a, LEN * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dev_b, LEN * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dev_c, LEN * sizeof(int)));
    CUDA_CHECK(hipMemcpy(dev_a, host_a, LEN * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_b, host_b, LEN * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dev_c, 0, LEN * sizeof(int))); // Set value by byte

    // add_kernel & result copy & print
    dim3 grid_dim(1, 1, 1);   // gridDim.x,  gridDim.y,  gridDim.z (always 1)
    dim3 block_dim(16, 1, 1); // blockDim.x, blockDim.y, blockDim.z
    add_kernel<int><<<grid_dim, block_dim>>>(LEN, dev_a, dev_b, dev_c);
    //add_kernel<<<1, 16>>>(LEN, dev_a, dev_b, dev_c); // Set gridDim.x & blockDim.x
    CUDA_CHECK(hipMemcpy(host_c, dev_c, LEN * sizeof(int), hipMemcpyDeviceToHost));
    VECTOR_PRINT("add_kernel results", host_c, LEN);

    // Free gpu memory & free cpu memory
    CUDA_CHECK(hipFree(dev_a));
    CUDA_CHECK(hipFree(dev_b));
    CUDA_CHECK(hipFree(dev_c));
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;
    return 0;
}
