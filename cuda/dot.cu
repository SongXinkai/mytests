#include <hip/hip_runtime.h>
#include <iostream>

using std::cin;
using std::cout;
using std::endl;

#define CUDA_CHECK(x) \
    { hipError_t cuda_error = x; \
        if (cuda_error != hipSuccess) \
            cout << "cudaError_t: " << cuda_error << " != 0 " \
                 << hipGetErrorString(cuda_error) << endl; \
    }

#define VECTOR_PRINT(head_str, vec, len) \
    cout << head_str << ": {"; \
    for (int i = 0; i < len - 1; ++i){ \
        cout << vec[i] << ", "; \
    } \
    cout << vec[len - 1] << "}" << endl;

#define LEN 8192
#define BLOCKDIM 256
#define GRIDDIM 8

// kernel function
__global__ void dot_kernel(int N, int *a, int *b, int *c ) { 
    __shared__ int cache[BLOCKDIM]; // 512 >= thread_id in block
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // thread_id in grid
    int cacheIndex = threadIdx.x; // thread_id in block
    // thread_num in grid

    int temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;//if blockDim == 1, then result = the sum of cache[].

    //同步
    __syncthreads();//make sure that all the threads in a block finish the procedure above

    //规约求和
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }

        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main() {
    // host memory malloc & initial
    int* host_a = new int[LEN];
    int* host_b = new int[LEN];
    int* host_c = new int[GRIDDIM];
    for (int i = 0; i < LEN; ++i) {
        host_a[i] = 1;
        host_b[i] = i;
    }
    for (int i = 0; i < GRIDDIM; ++i) {
        host_c[i] = 0;
    }

    // GPU device start
    int device_id = 1;
    CUDA_CHECK(hipSetDevice(device_id));
    cout << "Using GPU " << device_id << "." << endl;

    // cudaMalloc & cudaMemcpy & cudaMemset
    int* dev_a;
    int* dev_b;
    int* dev_c;
    CUDA_CHECK(hipMalloc((void**)&dev_a, LEN * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dev_b, LEN * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dev_c, GRIDDIM * sizeof(int)));
    CUDA_CHECK(hipMemcpy(dev_a, host_a, LEN * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_b, host_b, LEN * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dev_c, 0, GRIDDIM * sizeof(int)));

    // add_kernel & result copy & print
    dim3 grid_dim(GRIDDIM, 1, 1);   // gridDim.x,  gridDim.y,  gridDim.z
    dim3 block_dim(BLOCKDIM, 1, 1); // blockDim.x, blockDim.y, blockDim.z
    const int blocksPerGrid = grid_dim.x * grid_dim.y * grid_dim.z;
    dot_kernel<<<grid_dim, block_dim>>>(LEN, dev_a, dev_b, dev_c);
    CUDA_CHECK(hipMemcpy(host_c, dev_c, GRIDDIM * sizeof(int), hipMemcpyDeviceToHost));
    int c = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        cout << "host_c[" << i << "]: " << host_c[i] << endl;
        c += host_c[i];
    }
    cout << "dot_kernel results: " << c << endl;        

    // Free gpu memory & free cpu memory
    CUDA_CHECK(hipFree(dev_a));
    CUDA_CHECK(hipFree(dev_b));
    CUDA_CHECK(hipFree(dev_c));
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;
    return 0;
}
