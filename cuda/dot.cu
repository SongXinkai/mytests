#include <hip/hip_runtime.h>
#include <iostream>

using std::cin;
using std::cout;
using std::endl;

#define CUDA_CHECK(x) \
    { hipError_t cuda_error = x; \
        if (cuda_error != hipSuccess) \
            cout << "cudaError_t: " << cuda_error << " != 0 " \
                 << hipGetErrorString(cuda_error) << endl; \
    }

#define VECTOR_PRINT(head_str, vec, len) \
    cout << head_str << ": {"; \
    for (int i = 0; i < len - 1; ++i){ \
        cout << vec[i] << ", "; \
    } \
    cout << vec[len - 1] << "}" << endl;

#define LEN 16

// kernel function
__global__ void dot_kernel(int N, int *a, int *b, int *c ) { 
    __shared__ int cache[512];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    int temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;//if blockDim == 1, then result = the sum of cache[].

    //同步
    __syncthreads();//make sure that all the threads in a block finish the procedure above

    //规约求和
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }

        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main() {
    // host memory malloc & initial
    int* host_a = new int[LEN];
    int* host_b = new int[LEN];
    int* host_c = new int[LEN];
    for (int i = 0; i < LEN; ++i) {
        host_a[i] = 1;
        host_b[i] = i;
        host_c[i] = 0;
    }

    // GPU device start
    int device_id = 1;
    CUDA_CHECK(hipSetDevice(device_id));
    cout << "Using GPU " << device_id << "." << endl;

    // cudaMalloc & cudaMemcpy & cudaMemset
    int* dev_a;
    int* dev_b;
    int* dev_c;
    CUDA_CHECK(hipMalloc((void**)&dev_a, LEN * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dev_b, LEN * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&dev_c, LEN * sizeof(int)));
    CUDA_CHECK(hipMemcpy(dev_a, host_a, LEN * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_b, host_b, LEN * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dev_c, 0, LEN * sizeof(int)));

    // add_kernel & result copy & print
    dim3 grid_dim(1, 1, 1);   // gridDim.x,  gridDim.y,  gridDim.z
    dim3 block_dim(16, 1, 1); // blockDim.x, blockDim.y, blockDim.z
    const int blocksPerGrid = grid_dim.x * grid_dim.y * grid_dim.z;
    dot_kernel<<<grid_dim, block_dim>>>(LEN, dev_a, dev_b, dev_c);
    CUDA_CHECK(hipMemcpy(host_c, dev_c, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));
    int c = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        c += host_c[i];
    }
    cout << "dot_kernel results: " << c << endl;        

    // Free gpu memory & free cpu memory
    CUDA_CHECK(hipFree(dev_a));
    CUDA_CHECK(hipFree(dev_b));
    CUDA_CHECK(hipFree(dev_c));
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;
    return 0;
}
