#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::endl;

#define CUDA_CHECK(x) \
    { hipError_t cuda_error = x; \
        if (cuda_error != hipSuccess) \
            cout << "cudaError_t: " << cuda_error << " != 0 " \
                 << hipGetErrorString(cuda_error) << endl; \
    }

#define VECTOR_PRINT(head_str, vec, len) \
    cout << head_str << ": {"; \
    for (int i = 0; i < len - 1; ++i){ \
        cout << vec[i] << ", "; \
    } \
    cout << vec[len - 1] << "}" << endl;

#define LEN 32

// kernel functions
template<typename Dtype>
__global__ void set_kernel(const int N, Dtype* c){
    int bid = blockIdx.y * gridDim.x + blockIdx.x;
    int tid = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z * blockDim.y + bid * blockDim.x * blockDim.y * blockDim.z;
    c[tid] = blockIdx.x * 10000
         + blockIdx.y * 1000
         + threadIdx.x * 100
         + threadIdx.y * 10
         + threadIdx.z * 1;
}

int main(){
    // host memory malloc & initial
    int* host_a = new int[LEN];
    for (int i = 0; i < LEN; ++i){
        host_a[i] = 0;
    }
    
    // GPU device start
    int device_id = 1;
    CUDA_CHECK(hipSetDevice(device_id));
    cout << "Using GPU " << device_id << "." << endl;
    
    // cudaMalloc & cudaMemcpy & cudaMemset
    int* dev_a;
    CUDA_CHECK(hipMalloc((void**)&dev_a, LEN * sizeof(int)));
    CUDA_CHECK(hipMemcpy(dev_a, host_a, LEN * sizeof(int), hipMemcpyHostToDevice));

    // set_kernel & result copy & print
    dim3 grid_dim(2, 2, 1);   // gridDim.x,  gridDim.y,  gridDim.z (always 1)
    dim3 block_dim(2, 2, 2); // blockDim.x, blockDim.y, blockDim.z
    set_kernel<int><<<grid_dim, block_dim>>>(LEN, dev_a);
    CUDA_CHECK(hipMemcpy(host_a, dev_a, LEN * sizeof(int), hipMemcpyDeviceToHost));
    VECTOR_PRINT("set_kernel results", host_a, LEN);

    // Free gpu memory & free cpu memory
    CUDA_CHECK(hipFree(dev_a));
    delete[] host_a;
    return 0;
}
