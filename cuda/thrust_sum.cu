#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include <algorithm>

using thrust::sort;
using thrust::device_ptr;
using std::cout;
using std::endl;

#define CUDA_CHECK(x) \
    { hipError_t cuda_error = x; \
        if (cuda_error != hipSuccess) \
            cout << "hipError_t: " << cuda_error << " != 0 " \
                 << hipGetErrorString(cuda_error) << endl; \
    }

#define VECTOR_PRINT(head_str, vec, LEN) \
    cout << head_str << ": {"; \
    for (int i = 0; i < LEN - 1; ++i){ \
        cout << vec[i] << ", "; \
    } \
    cout << vec[LEN - 1] << "}" << endl;

#define LEN 32

int main(){
    // host memory malloc & initial
    float* host_a = new float[LEN];
    for (int i = 0; i < LEN; ++i){
        host_a[i] = LEN - i;
    }
    
    // GPU device start
    int device_id = 1;
    CUDA_CHECK(hipSetDevice(device_id));
    cout << "Using GPU " << device_id << "." << endl;
    
    // hipMalloc & hipMemcpy & hipMemset
    float* dev_a;
    CUDA_CHECK(hipMalloc((void**)&dev_a, LEN * sizeof(float)));
    CUDA_CHECK(hipMemcpy(dev_a, host_a,  LEN * sizeof(float), hipMemcpyHostToDevice));

    // thrust reduction max
    device_ptr<float> dp(dev_a);
    thrust::plus<float> add_op;
    cout << "sum: " << thrust::reduce(dp, dp + LEN, float(100), add_op) << endl;

    // Free gpu memory & free cpu memory
    CUDA_CHECK(hipFree(dev_a));
    delete[] host_a;
    return 0;
}
