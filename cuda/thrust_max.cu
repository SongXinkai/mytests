#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include <algorithm>

using thrust::sort;
using thrust::device_ptr;
using std::cout;
using std::endl;

#define CUDA_CHECK(x) \
    { hipError_t cuda_error = x; \
        if (cuda_error != hipSuccess) \
            cout << "hipError_t: " << cuda_error << " != 0 " \
                 << hipGetErrorString(cuda_error) << endl; \
    }

#define VECTOR_PRINT(head_str, vec, LEN) \
    cout << head_str << ": {"; \
    for (int i = 0; i < LEN - 1; ++i){ \
        cout << vec[i] << ", "; \
    } \
    cout << vec[LEN - 1] << "}" << endl;

#define LEN 32

template <typename Dtype>
struct MAX_OP{
    __host__ __device__ 
    Dtype operator() (const Dtype& x, const Dtype& y) const {
        return x > y ? x : y;
    }
};

int main(){
    // host memory malloc & initial
    float* host_a = new float[LEN];
    for (int i = 0; i < LEN; ++i){
        host_a[i] = LEN - i;
    }
    
    // GPU device start
    int device_id = 1;
    CUDA_CHECK(hipSetDevice(device_id));
    cout << "Using GPU " << device_id << "." << endl;
    
    // hipMalloc & hipMemcpy & hipMemset
    float* dev_a;
    CUDA_CHECK(hipMalloc((void**)&dev_a, LEN * sizeof(float)));
    CUDA_CHECK(hipMemcpy(dev_a, host_a,  LEN * sizeof(float), hipMemcpyHostToDevice));

    // thrust reduction max
    device_ptr<float> dp(dev_a);
    MAX_OP<float> max_op;
    cout << "max: " << thrust::reduce(dp, dp + LEN, float(0), max_op) << endl;

    // Free gpu memory & free cpu memory
    CUDA_CHECK(hipFree(dev_a));
    delete[] host_a;
    return 0;
}
