#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include <algorithm>

using thrust::sort;
using thrust::device_ptr;
using std::cout;
using std::endl;

#define CUDA_CHECK(x) \
    { hipError_t cuda_error = x; \
        if (cuda_error != hipSuccess) \
            cout << "hipError_t: " << cuda_error << " != 0 " \
                 << hipGetErrorString(cuda_error) << endl; \
    }

#define VECTOR_PRINT(head_str, vec, LEN) \
    cout << head_str << ": {"; \
    for (int i = 0; i < LEN - 1; ++i){ \
        cout << vec[i] << ", "; \
    } \
    cout << vec[LEN - 1] << "}" << endl;

#define LEN 32

int main(){
    // host memory malloc & initial
    float* host_a = new float[LEN];
    float* host_b = new float[LEN];
    for (int i = 0; i < LEN; ++i){
        host_a[i] = LEN - i;
        host_b[i] = LEN - i;
    }
    
    // GPU device start
    int device_id = 1;
    CUDA_CHECK(hipSetDevice(device_id));
    cout << "Using GPU " << device_id << "." << endl;
    
    // hipMalloc & hipMemcpy & hipMemset
    float* dev_a;
    CUDA_CHECK(hipMalloc((void**)&dev_a, LEN * sizeof(float)));
    CUDA_CHECK(hipMemcpy(dev_a, host_a,  LEN * sizeof(float), hipMemcpyHostToDevice));

    // thrust device sort
    device_ptr<float> dp(dev_a);
    sort(dp, dp + LEN);
    CUDA_CHECK(hipMemcpy(host_a, dev_a, LEN * sizeof(float), hipMemcpyDeviceToHost));
    VECTOR_PRINT("thrust sort", host_a, LEN);

    // std::sort
    for (int i = 0; i < LEN; ++i){
        host_b[i] = float(LEN) - float(i);
    }
    std::sort(host_b, host_b+ LEN);
    VECTOR_PRINT("std sort", host_b, LEN);

    // Free gpu memory & free cpu memory
    CUDA_CHECK(hipFree(dev_a));
    delete[] host_a;
    delete[] host_b;
    return 0;
}
